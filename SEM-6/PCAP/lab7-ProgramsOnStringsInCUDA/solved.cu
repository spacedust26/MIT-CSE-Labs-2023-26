// Solved Exercise - A CUDA program which takes a string as input and determines the number of occurances of a character 'a' in the string. This program uses atomicAdd() function.

#include "hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#define N 256

__global__ void CUDACount(char *A, unsigned int *d_count){
    int i = threadIdx.x;
    if(A[i] == 'a') atomicAdd(d_count, 1);
}

int main(){
    char A[N];
    char *d_A;
    unsigned int *count = 0, *d_count, *result;

    printf("Enter a string: ");
    scanf("%s", A);

    result = (unsigned int*)malloc(sizeof(unsigned int)); 

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_A, N * sizeof(char)); 
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    hipMemcpy(d_count, count, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess) printf("CUDA Error1: %s\n", hipGetErrorString(error));

    CUDACount<<<1, strlen(A)>>>(d_A, d_count);

    error = hipGetLastError();
    if(error != hipSuccess) printf("CUDA Error2: %s\n", hipGetErrorString(error));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurrences of 'a' = %u\n", *result);
    printf("Time taken = %f ms\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_count);
    free(result);  
    return 0;
}
