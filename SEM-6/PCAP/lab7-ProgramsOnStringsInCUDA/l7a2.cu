// Additional Exercise 2 - Write a CUDA program that takes a string Sin as input and one integer value N and produces an output string, Sout, in parallel by concatenating niput string Sin, N times as shown below.
// Input : Sin = "Hello", N = 3
// Output : Sout = "HelloHelloHello"

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__ void concatenateStrings(char *sin, char *sout, int sinLength, int n) {
    int tid = threadIdx.x;
    int index = tid * sinLength;  
    if (tid < n) {
        for (int i = 0; i < sinLength; i++) {
            sout[index + i] = sin[i];
        }
    }
}

int main() {
    char sin[100] = "Hello";   
    int n = 3;            
    char sout[100];         

    int sinLength = strlen(sin); 
    char *d_sin, *d_sout;
    hipMalloc((void**)&d_sin, sinLength * sizeof(char));
    hipMalloc((void**)&d_sout, sinLength * n * sizeof(char));
    hipMemcpy(d_sin, sin, sinLength * sizeof(char), hipMemcpyHostToDevice);

    concatenateStrings<<<1, n>>>(d_sin, d_sout, sinLength, n);
    hipMemcpy(sout, d_sout, sinLength * n * sizeof(char), hipMemcpyDeviceToHost);
    printf("Output string: %s\n", sout);
    hipFree(d_sin);
    hipFree(d_sout);
    return 0;
}
