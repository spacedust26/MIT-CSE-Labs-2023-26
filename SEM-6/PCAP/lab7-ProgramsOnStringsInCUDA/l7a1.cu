// additional Exerice 1 - write a CUDA program which reads a string consisting of N words and reverses each word of it parallel

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__ void reverseFunction(char *arr, int *startindex, int *endindex){
    int tid = threadIdx.x;
    int start = startindex[tid];
    int end = endindex[tid];

    while(start < end){
        char temp = arr[start];
        arr[start] = arr[end];
        arr[end] = temp;
        start++;end--;
    }
}

int main(){
    char str[100] = "hello world hi there";
    char res[100];
    int n = strlen(str);
    int startIndex[100], endIndex[100], totalWords=0;
    int i=0,j=0,k=0;
    while(j < n){
        if(str[j] == ' '){
            startIndex[k] = i;
            endIndex[k++] = j-1;
            i = j + 1;
            totalWords++;
        }
        j++;
    }
    startIndex[k] = i;
    endIndex[k] = j-1;
    totalWords++;

    char *d_str;
    int *d_startIndex, *d_endIndex;
    hipMalloc((void**)&d_str, n * sizeof(char));
    hipMalloc((void**)&d_startIndex, k * sizeof(int));
    hipMalloc((void**)&d_endIndex, k * sizeof(int));
    hipMemcpy(d_str, str, n * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_startIndex, startIndex, k * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_endIndex, endIndex, k * sizeof(int), hipMemcpyHostToDevice);

    reverseFunction<<<1, totalWords>>>(d_str, d_startIndex, d_endIndex);
    hipMemcpy(res, d_str, n * sizeof(char), hipMemcpyDeviceToHost);
    for(int i=0;i<n;i++) printf("%c", res[i]);

    hipFree(d_str);
    hipFree(d_startIndex);
    hipFree(d_endIndex);
    return 0;
}