// Write a CUDA program that reads a string S and produces the string RS as follows:
// Input string S: PCAP Output string RS: PCAPPCAPCP

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__ void pattern(char* s,int len,char* rs){
    int gtid = threadIdx.x;
    int pos = (gtid *(2*len + (gtid-1)*-1))/2; 
    int finlen = len - gtid;
    for(int i=0;i<finlen;i++){
        rs[pos++]=s[i];
    }

}

int main(){
    char s[20];
    printf("Input string s: ");
    scanf("%s",s);

    int len = strlen(s);
    int rslen = len*(len+1)/2;
    char rs[rslen];
    char * d_s;
    char * d_rs;

    hipMalloc((void**)&d_s,len*sizeof(char));
    hipMalloc((void**)&d_rs,rslen*sizeof(char));

    hipMemcpy(d_s,s,len*sizeof(char),hipMemcpyHostToDevice);

    pattern<<<1,len>>>(d_s,len,d_rs);

    hipMemcpy(rs,d_rs,rslen*sizeof(char),hipMemcpyDeviceToHost);
    rs[rslen]='\0';
    printf("Output string rs: %s\n",rs);

    hipFree(d_s);
    hipFree(d_rs);
    return 0;
}