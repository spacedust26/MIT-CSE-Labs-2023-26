// Write a CUDA program that reads a string S and produces the string RS as follows:
// Input string S: PCAP Output string RS: PCAPPCAPCP

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__ void buildRS(char* s, char* rs, int len) {
    int i = threadIdx.x;
    // Calculate starting offset in the output string
    int offset = 0;
    for (int j = 0; j < i; j++) {
        offset += len - j;
    }
    // Copy first (len - i) characters of s to rs at offset
    for (int j = 0; j < len - i; j++) {
        rs[offset + j] = s[j];
    }
}

int main() {
    char s[100];
    printf("Enter string S: ");
    scanf("%s", s);
    int len = strlen(s);

    int rs_len = len * (len + 1) / 2;  // total output size
    char* rs = (char*)malloc((rs_len + 1) * sizeof(char));  // +1 for '\0'
    rs[rs_len] = '\0';

    // Allocate device memory
    char *d_s, *d_rs;
    hipMalloc((void**)&d_s, len * sizeof(char));
    hipMalloc((void**)&d_rs, rs_len * sizeof(char));

    hipMemcpy(d_s, s, len * sizeof(char), hipMemcpyHostToDevice);

    // Launch kernel
    buildRS<<<1, len>>>(d_s, d_rs, len);
    hipDeviceSynchronize();

    hipMemcpy(rs, d_rs, rs_len * sizeof(char), hipMemcpyDeviceToHost);

    printf("Output string RS: %s\n", rs);

    hipFree(d_s);
    hipFree(d_rs);
    free(rs);

    return 0;
}