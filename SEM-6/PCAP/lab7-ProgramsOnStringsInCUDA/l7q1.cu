//Write a program in CUDA to count the number of times a given word is repeated in a sentence. (Use atomic functions)

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__  void  wordMatch(char* str,int len,char* key,int keylen,int * wordindex,int * count){
    int id = threadIdx.x;
    int i = wordindex[id];
    int j = 0;
    int match = 1;
    while(i<len && j<keylen){
        if(str[i]!=key[j]){
            match = 0;
            break;
        }
        i++;
        j++;
    }
    if(j==keylen && match){
        atomicAdd(count,1);
    }
}

int main(){
    char str[100];
    printf("Enter sentence: ");
    scanf("%[^\n]c",str);
    int len = strlen(str);
    
    char key[20];
    printf("Enter key: ");
    scanf(" %[^\n]c",key);
    int keylen = strlen(key);
 
    int wordcount=0;
    for(int i=0;i<len;i++){
        if(str[i]==' ' || i == len -1)
            wordcount++;
    }
    
    int wordindex[wordcount];
    int j=0;
    int k=1;
    wordindex[j++]=0;
    for(int i=0;i<len;i++){
        if(str[i]==' '){
            wordindex[j++]=i+1;
            k++;
        }    
        if(k==wordcount) break;
    } 

    int count = 0;
    char * d_str, * d_key;
    int * d_count,* d_wordindex;

    hipMalloc((void**)&d_str,len*sizeof(char));
    hipMalloc((void**)&d_key,keylen*sizeof(char));
    hipMalloc((void**)&d_count,sizeof(int));
    hipMalloc((void**)&d_wordindex,wordcount*sizeof(int));

    hipMemcpy(d_str,str,len*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_key,key,keylen*sizeof(char),hipMemcpyHostToDevice);
    hipMemcpy(d_wordindex,wordindex,wordcount*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);

    wordMatch<<<1,wordcount>>>(d_str,len,d_key,keylen,d_wordindex,d_count);

    hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);

    printf("Occurrances: %d\n",count);

    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_count);
    hipFree(d_wordindex);
    return 0;

}