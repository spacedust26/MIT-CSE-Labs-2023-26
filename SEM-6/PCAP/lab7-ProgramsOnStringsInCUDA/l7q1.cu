//Exercise 1 - Write a program in CUDA to count the number of times a given word is repeated in a sentence. (Use atomic functions)
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__device__ bool match_word(char* str, int idx, char* key, int keylen) {
    for (int i = 0; i < keylen; i++) {
        if (str[idx + i] != key[i]) {
            return false;
        }
    }
    return true;
}

__global__ void wordMatch(char* str, int len, char* key, int keylen, int *count) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < len - keylen + 1) {
        if (match_word(str, id, key, keylen)) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    char str[100] = "hello there my name is hello nice to meet you hello";
    // printf("Enter sentence: ");
    // scanf("%[^\n]%*c", str);
    int len = strlen(str);
    
    char key[20] = "hello";
    // printf("Enter key: ");
    // scanf("%[^\n]%*c", key);
    int keylen = strlen(key);
    
    // Prepare for CUDA
    int count = 0;
    char *d_str, *d_key;
    int *d_count;
    
    hipMalloc((void**)&d_str, len * sizeof(char));
    hipMalloc((void**)&d_key, keylen * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));
    
    hipMemcpy(d_str, str, len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_key, key, keylen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to count word occurrences
    wordMatch<<<ceil((float)len/256), 256>>>(d_str, len, d_key, keylen, d_count);
    hipDeviceSynchronize();
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    printf("Occurrences of the word '%s': %d\n", key, count);

    // Cleanup
    hipFree(d_str);
    hipFree(d_key);
    hipFree(d_count);
    return 0;
}