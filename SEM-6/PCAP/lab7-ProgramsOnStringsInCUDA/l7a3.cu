// Additional Exercise 3 - Write a CUDA program which reads a string Sin and produces an output string T as shown below.
// Input: Sin = "Hai"
// Output: T = "Haaiii"

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>

__global__ void transformString(char *sin, char *t, int sinLength) {
    int tid = threadIdx.x;
    if (tid < sinLength) {
        char ch = sin[tid];
        int startIndex = 0;
        for (int i = 0; i < tid; i++) {
            startIndex += (i + 1); 
        }
        for (int i = 0; i <= tid; i++) {
            t[startIndex + i] = ch;
        }
    }
}

int main() {
    char sin[100] = "Hai"; 
    int sinLength = strlen(sin);
    char t[100]; 
    int totalLength = (sinLength * (sinLength+1)) / 2;

    char *d_sin, *d_t;
    hipMalloc((void**)&d_sin, sinLength * sizeof(char));
    hipMalloc((void**)&d_t, totalLength * sizeof(char));
    hipMemcpy(d_sin, sin, sinLength * sizeof(char), hipMemcpyHostToDevice);

    transformString<<<1, sinLength>>>(d_sin, d_t, sinLength);
    hipMemcpy(t, d_t, totalLength * sizeof(char), hipMemcpyDeviceToHost);
    t[totalLength] = '\0';
    printf("Output string: %s\n", t);

    hipFree(d_sin);
    hipFree(d_t);
    return 0;
}
