// Lab Exercise 1
// Write a program in CUDA to add two Matrices for the following specifications
// a. Each row of the resultant matrix to be computed by one thread
// b. Each column of resultant matrix to be computed by one thread
// c. Ecah element of resultant matrix to be computed by one thread

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void addMatrixByRow(int *a, int *b, int *c, int m, int n){
    int row = threadIdx.x;
    if(row < m){
        for(int col = 0; col < n; col++){
            c[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void addMatrixByColumn(int *a, int *b, int *c, int m, int n){
    int col = threadIdx.x;
    if(col < n){
        for(int row = 0; row < m; row++){
            c[row * n + col] = a[row * n + col] + b[row * n + col];
        }
    }
}

__global__ void addMatrixByElement(int *a, int *b, int *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        c[row * n + col] = a[row * n + col] + b[row * n + col];
    }
}

void printMatrix(int *m, int m_rows, int n_cols){
    for(int i = 0; i < m_rows; i++){
        for(int j = 0; j < n_cols; j++){
            printf("%d\t", m[i*n_cols + j]);
        }
        printf("\n");
    }
}

int main(){
    int *h_a, *h_b, *h_c, m, n, *d_a, *d_b, *d_c;
    printf("Enter the number of rows (m) of the matrices: ");
    scanf("%d", &m);
    printf("Enter the number of columns (n) of the matrices: ");
    scanf("%d", &n);
    
    h_a = (int*)malloc(m * n * sizeof(int));
    h_b = (int*)malloc(m * n * sizeof(int));
    h_c = (int*)malloc(m * n * sizeof(int));
    
    printf("Enter the elements for matrix A: ");
    for(int i = 0; i < m*n; i++) scanf("%d", &h_a[i]);
    
    printf("Enter the elements for matrix B: ");
    for(int i = 0; i < m*n; i++) scanf("%d", &h_b[i]);

    hipMalloc((void**)&d_a, m * n * sizeof(int));
    hipMalloc((void**)&d_b, m * n * sizeof(int));
    hipMalloc((void**)&d_c, m * n * sizeof(int));

    hipMemcpy(d_a, h_a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, m * n * sizeof(int), hipMemcpyHostToDevice);

    // Row-wise computation
    addMatrixByRow<<<1, m>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant matrix after row-wise computation:\n");
    printMatrix(h_c, m, n);

    // Column-wise computation
    addMatrixByColumn<<<1, n>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant matrix after column-wise computation:\n");
    printMatrix(h_c, m, n);

    // Element-wise computation
    dim3 threadsPerBlock(16, 16); 
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x, (m + threadsPerBlock.y - 1) / threadsPerBlock.y);                   
    addMatrixByElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant matrix after element-wise computation:\n");
    printMatrix(h_c, m, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
