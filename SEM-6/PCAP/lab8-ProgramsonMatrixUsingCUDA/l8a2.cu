// Additional Exercise 2 - Write a CUDA program to read a matrix A of size N X N. It replaces the principal diagonal elements with zero. Elements above the principal diagonal by thier factorial and elements below the principal diagonal by their sum of digits.

#include <stdio.h>
#include "hip/hip_runtime.h"


#define N 3

__device__ int factorial(int n) {
    int f = 1;
    for (int i = 2; i <= n; i++) {
        f *= i;
    }
    return f;
}
__device__ int sumOfDigits(int n) {
    int sum = 0;
    while (n != 0) {
        sum += n % 10;
        n /= 10;
    }
    return sum;
}
__global__ void processMatrix(int *A, int *B, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n || col >= n) return;
    int idx = row * n + col;
    int val = A[idx];
    if (row == col) B[idx] = 0;
    else if (row < col) B[idx] = factorial(val);
    else B[idx] = sumOfDigits(val);
}

int main() {
    int A[N][N] = {
        {1, 2, 3},
        {4, 5, 6},
        {7, 8, 9}
    };
    int B[N][N];
    int size = N * N * sizeof(int);
    int *d_A, *d_B;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (N + 15) / 16);
    processMatrix<<<gridDim, blockDim>>>(d_A, d_B, N);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    printf("Resulting Matrix B:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d\t", B[i][j]);
        }
        printf("\n");
    }
    hipFree(d_A);
    hipFree(d_B);
    return 0;
}
