// Lab Exercise 2
// Write a program in CUDA to multiply two matrices for the following specifictaions:
// a. Each row of resultant matrix to be computed by one thread
// b. Each column of resultant matrix to be computed by one thread
// c. Each element of resultant matrix to be computed by one thread

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void mulMatrixByRow(int *a, int *b, int *c, int wa, int wb) {
    int row = threadIdx.x;
    for (int col = 0; col < wb; col++) {
        c[row * wb + col] = 0;
        for (int k = 0; k < wa; k++) {
            c[row * wb + col] += a[row * wa + k] * b[k * wb + col];
        }
    }
}

__global__ void mulMatrixByColumn(int *a, int *b, int *c, int ha, int wa) {
    int col = threadIdx.x;
    for (int row = 0; row < ha; row++) {
        c[row * wa + col] = 0;
        for (int k = 0; k < wa; k++) {
            c[row * wa + col] += a[row * wa + k] * b[k * wa + col];
        }
    }
}

__global__ void mulMatrixByElement(int *a, int *b, int *c, int ha, int wb, int wa) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < ha && col < wb) {
        c[row * wb + col] = 0;
        for (int k = 0; k < wa; k++) {
            c[row * wb + col] += a[row * wa + k] * b[k * wb + col];
        }
    }
}

void printMatrix(int *m, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%d\t", m[i * col + j]);
        }
        printf("\n");
    }
}

int main() {
    int *h_a, *h_b, *h_c, *d_a, *d_b, *d_c;
    int ha, wa, hb, wb;
    
    printf("Enter the dimensions of matrix A: ");
    scanf("%d %d", &ha, &wa);
    printf("Enter the dimensions of matrix B: ");
    scanf("%d %d", &hb, &wb);

    if (wa != hb) {
        printf("Matrix dimensions are incompatible for multiplication.\n");
        return -1;
    }

    h_a = (int*)malloc(ha * wa * sizeof(int));
    h_b = (int*)malloc(hb * wb * sizeof(int));
    h_c = (int*)malloc(ha * wb * sizeof(int));

    printf("Enter the elements for matrix A: ");
    for (int i = 0; i < ha * wa; i++) scanf("%d", &h_a[i]);

    printf("Enter the elements for matrix B: ");
    for (int i = 0; i < hb * wb; i++) scanf("%d", &h_b[i]);

    hipMalloc((void**)&d_a, ha * wa * sizeof(int));
    hipMalloc((void**)&d_b, hb * wb * sizeof(int));
    hipMalloc((void**)&d_c, ha * wb * sizeof(int));

    hipMemcpy(d_a, h_a, ha * wa * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, hb * wb * sizeof(int), hipMemcpyHostToDevice);

    // Row-wise matrix multiplication
    mulMatrixByRow<<<1, ha>>>(d_a, d_b, d_c, wa, wb);
    hipMemcpy(h_c, d_c, ha * wb * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant matrix after row wise computation:\n");
    printMatrix(h_c, ha, wb);

    // Column-wise matrix multiplication
    mulMatrixByColumn<<<1, wb>>>(d_a, d_b, d_c, ha, wa);
    hipMemcpy(h_c, d_c, ha * wb * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant matrix after column wise computation:\n");
    printMatrix(h_c, ha, wb);

    // Element-wise matrix multiplication
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((wb + threadsPerBlock.x - 1) / threadsPerBlock.x, (ha + threadsPerBlock.y - 1) / threadsPerBlock.y);
    mulMatrixByElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, ha, wb, wa);
    hipMemcpy(h_c, d_c, ha * wb * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant matrix after element wise computation:\n");
    printMatrix(h_c, ha, wb);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
