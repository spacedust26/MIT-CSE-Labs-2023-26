// Solved Exercise - Write a program in CUDA to find the transpose of a matrix in parallel

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void transpose(int* input, int* output, int rows, int cols) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // column index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row index
    if (x < cols && y < rows) {
        output[x * rows + y] = input[y * cols + x];
    }
}

int main() {
    int rows, cols;
    printf("Enter number of rows: ");
    scanf("%d", &rows);
    printf("Enter number of columns: ");
    scanf("%d", &cols);
    int size = rows * cols;
    int* h_input = (int*)malloc(size * sizeof(int));
    int* h_output = (int*)malloc(size * sizeof(int));
    printf("Enter %d elements (row-wise):\n", size);
    for (int i = 0; i < size; i++) {
        scanf("%d", &h_input[i]);
    }

    int *d_input, *d_output;
    hipMalloc((void**)&d_input, size * sizeof(int));
    hipMalloc((void**)&d_output, size * sizeof(int));
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((cols + 15) / 16, (rows + 15) / 16);
    transpose<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, rows, cols);
    hipMemcpy(h_output, d_output, size * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nTransposed Matrix (%dx%d):\n", cols, rows);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", h_output[i * cols + j]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}
