// Additional Exercise 1 - Write a CUDA program that reads a M X N matrix A and produces a resultant matrix B of same size as follows: Replcae all the even numbered matrix elements with their row sum and odd numbered elements with their column sum.
// example: A = 1 2 3           B =  5 6 9
//              4 5 6                15 7 15

#include <stdio.h>
#include "hip/hip_runtime.h"


#define M 2
#define N 3

__global__ void processMatrix(int *A, int *B, int *rowSum, int *colSum, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        if (A[idx] % 2 == 0)
            B[idx] = rowSum[row];
        else
            B[idx] = colSum[col];
    }
}

int main() {
    int A[M][N] = {
        {1, 2, 3},
        {4, 5, 6}
    };
    int B[M][N];
    int rowSum[M] = {0};
    int colSum[N] = {0};

    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++) {
            rowSum[i] += A[i][j];
            colSum[j] += A[i][j];
        }

    int *d_A, *d_B, *d_rowSum, *d_colSum;
    size_t sizeMatrix = M * N * sizeof(int);
    hipMalloc((void**)&d_A, sizeMatrix);
    hipMalloc((void**)&d_B, sizeMatrix);
    hipMalloc((void**)&d_rowSum, M * sizeof(int));
    hipMalloc((void**)&d_colSum, N * sizeof(int));

    hipMemcpy(d_A, A, sizeMatrix, hipMemcpyHostToDevice);
    hipMemcpy(d_rowSum, rowSum, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colSum, colSum, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + 15) / 16, (M + 15) / 16);

    processMatrix<<<gridDim, blockDim>>>(d_A, d_B, d_rowSum, d_colSum, M, N);

    hipMemcpy(B, d_B, sizeMatrix, hipMemcpyDeviceToHost);

    printf("Resulting Matrix B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
}
