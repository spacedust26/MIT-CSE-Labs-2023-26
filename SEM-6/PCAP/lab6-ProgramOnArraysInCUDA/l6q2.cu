// Exercise 2 - Write a program in CUDA to perform selection sort in parallel.

#include "hip/hip_runtime.h"

#include<stdio.h>

//Each thread calculates the final position of its element by counting how many elements are less than it.
__global__ void parallelSelectionSort(int*d_a,int*d_c,int s){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int k=0;
    for(int j=0;j<s;j++){
        if((d_a[j]<d_a[i])||(d_a[j]==d_a[i]&&j>i))
            k++;
    }
    d_c[k]=d_a[i];
}

int main(){
    int n;
    printf("Enter number of elements: ");
    scanf("%d",&n);
    int a[n],c[n];
    int *d_a,*d_c;

    hipMalloc((void **)&d_a,n*sizeof(int));
    hipMalloc((void **)&d_c,n*sizeof(int));

    printf("Enter the elements: ");
    for(int i=0;i<n;i++) scanf("%d",&a[i]);
    
    hipMemcpy(d_a,a,n*sizeof(int),hipMemcpyHostToDevice);
    parallelSelectionSort<<<1, n>>>(d_a,d_c,n);
    hipMemcpy(c,d_c,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Sorted array:");
    for(int i=0;i<n;i++) printf("%d ",c[i]);
    hipFree(d_a);
    hipFree(d_c);
}