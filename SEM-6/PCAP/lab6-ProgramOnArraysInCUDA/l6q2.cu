#include "hip/hip_runtime.h"

#include<stdio.h>
__global__ void parallelSelectionSort(int*d_a,int*d_c,int s){
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int k=0;
    for(int j=0;j<s;j++){
        if((d_a[j]<d_a[i])||(d_a[j]==d_a[i]&&j>i))
            k++;
    }
    d_c[k]=d_a[i];
}

int main(){
    int n;
    printf("Enter number of elements: ");
    scanf("%d",&n);

    int a[n],c[n];
    int *d_a,*d_c;

    hipMalloc((void **)&d_a,n*sizeof(int));
    hipMalloc((void **)&d_c,n*sizeof(int));

    printf("Enter the elements: ");
    for(int i=0;i<n;i++)
        scanf("%d",&a[i]);
    
    hipMemcpy(d_a,a,n*sizeof(int),hipMemcpyHostToDevice);
    int threads_per_block = n;
    int num_blocks = 1;
    parallelSelectionSort<<<num_blocks, threads_per_block>>>(d_a,d_c,n);
    hipMemcpy(c,d_c,n*sizeof(int),hipMemcpyDeviceToHost);
    printf("Sorted array:");
    for(int i=0;i<n;i++)
        printf("%d ",c[i]);
    printf("\n");
    hipFree(d_a);
    hipFree(d_c);
}