// Additional Exercise 1 - write a program in CUDA which takes N integers as input. It converts these integers into their corresponding octal values and stores the result in another array in parallel

#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ void intToOctal(int num, char *result) {
    int index = 0;
    int octal[32];
    if (num == 0) {
        result[0] = '0';
        result[1] = '\0';
        return;
    }
    while (num > 0) {
        octal[index++] = num % 8;
        num /= 8;
    }
    for (int i = 0; i < index; i++) {
        result[i] = '0' + octal[index - i - 1];
    }
    result[index] = '\0'; 
}

__global__ void convertToOctal(int *d_input, char *d_output, int N, int maxLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        intToOctal(d_input[idx], &d_output[idx * maxLen]);
    }
}

int main() {
    int N;
    printf("Enter number of elements: ");
    scanf("%d", &N);
    int *h_input = new int[N];
    printf("Enter %d integers:\n", N);
    for (int i = 0; i < N; i++) scanf("%d", &h_input[i]);

    int *d_input;
    char *d_output;
    const int maxLen = 12;  // Max octal length for 32-bit int + null char
    char *h_output = new char[N * maxLen];
    hipMalloc(&d_input, sizeof(int) * N);
    hipMalloc(&d_output, sizeof(char) * N * maxLen);
    hipMemcpy(d_input, h_input, sizeof(int) * N, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    convertToOctal<<<blocks, threadsPerBlock>>>(d_input, d_output, N, maxLen);
    hipDeviceSynchronize();
    hipMemcpy(h_output, d_output, sizeof(char) * N * maxLen, hipMemcpyDeviceToHost);
    printf("\nOctal representations:\n");
    for (int i = 0; i < N; i++) {
        printf("%d -> %s\n", h_input[i], &h_output[i * maxLen]);
    }

    // Cleanup
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
