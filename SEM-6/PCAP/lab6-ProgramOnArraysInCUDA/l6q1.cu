// Exercise 1 - Write a program in CUDA which performs convolution operation on one-dimensional input array N of size width using a mask array M of size mask_width to produce the resultant one dimensional array P of size width

#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void convolution1d(int *n, int *m , int *p, int *width, int *mask_width){
    int i = threadIdx.x;
    int pvalue = 0;
    int n_start_point = i - (*mask_width / 2);
    for(int j=0; j<*mask_width;j++){
        if(n_start_point+j >= 0 && n_start_point+j < *width){
            pvalue += n[n_start_point + j] * m[j];
        }
    }
    p[i] = pvalue;
}

int main(){
    const int width = 10;
    const int mask_width = 5;
    int N[width] = {1,2,3,4,5,6,7,8,9,10};
    int P[width];
    int M[mask_width] = {2,3,5,7,1};

    int *d_N, *d_M, *d_P, *d_width, *d_mask_width;
    hipMalloc((void**)&d_N, width * sizeof(int));
    hipMalloc((void**)&d_P, width * sizeof(int));
    hipMalloc((void**)&d_M, mask_width * sizeof(int));
    hipMalloc((void**)&d_width, sizeof(int));
    hipMalloc((void**)&d_mask_width, sizeof(int));

    hipMemcpy(d_N, N, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask_width, &mask_width, sizeof(int), hipMemcpyHostToDevice);

    convolution1d<<<1, width>>>(d_N, d_M, d_P, d_width, d_mask_width);
    hipMemcpy(P, d_P, width*sizeof(width), hipMemcpyDeviceToHost);
    printf("Resultant array after convolution: ");
    for(int i=0;i<width;i++) printf("%d ", P[i]);

    hipFree(d_N);
    hipFree(d_P);
    hipFree(d_M);
    hipFree(d_width);
    hipFree(d_mask_width);
    return 0;
}
