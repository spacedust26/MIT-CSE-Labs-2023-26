#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>




__global__ void convulution_1D_basic_kernel(float *N, float *M, float *P,
                                             int Mask_Width, int Width) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  float Pvalue = 0;
  int start_point = i - (Mask_Width / 2);
  for (int j = 0; j < Mask_Width; j++) {
    if (start_point + j >= 0 && start_point + j < Width) {
      Pvalue += N[start_point + j] * M[j];
    }
  }
  if (i < Width) {
    P[i] = Pvalue;
  }
}

int main(int argc, char *argv[]) {
  float *A, *M, *P;
  float *d_A, *d_M, *d_P;
  int arrlen, mask_width;

  printf("Enter number of elements in the array:\n");
  scanf("%d", &arrlen);
  int inp_bytes = arrlen * sizeof(float);
  A = (float *)malloc(inp_bytes);
  P = (float *)malloc(inp_bytes);

  hipMalloc(&d_A, inp_bytes);
  hipMalloc(&d_P, inp_bytes);

  printf("Enter the elements:");
  for (int i = 0; i < arrlen; i++) {
    scanf("%f", &A[i]);
  }

  hipMemcpy(d_A, A, inp_bytes, hipMemcpyHostToDevice);

  printf("Enter number of elements in the mask:\n");
  scanf("%d", &mask_width);
  int mask_bytes = mask_width * sizeof(float);
  M = (float *)malloc(mask_bytes);
  hipMalloc(&d_M, mask_bytes);

  printf("Enter the elements in the mask:");
  for (int i = 0; i < mask_width; i++) {
    scanf("%f", &M[i]);
  }

  hipMemcpy(d_M, M, mask_bytes, hipMemcpyHostToDevice);

  int threads_per_block = arrlen; 
  int num_blocks = 1;

  convulution_1D_basic_kernel<<<num_blocks, threads_per_block>>>(
      d_A, d_M, d_P, mask_width, arrlen);


  hipMemcpy(P, d_P, inp_bytes, hipMemcpyDeviceToHost);

  for (int i = 0; i < arrlen; i++) {
    printf("%f\n", P[i]);
  }
  printf("\n");

  free(M);
  free(A);
  free(P);

  hipFree(d_M);
  hipFree(d_A);
  hipFree(d_P);

  return 0;
}
