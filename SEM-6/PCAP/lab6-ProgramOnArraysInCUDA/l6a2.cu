// Additional Exercise 2 - Write a CUDA program which takes N binary numbers as input and stores the one's complement of each element in another array in parallel

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LEN 32  // Max binary string length

__global__ void onesComplementKernel(char *d_input, char *d_output, int N, int maxLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int offset = idx * maxLen;
        for (int i = 0; i < maxLen; i++) {
            char bit = d_input[offset + i];
            if (bit == '\0') {
                d_output[offset + i] = '\0';  // End of string
                break;
            } else if (bit == '0') {
                d_output[offset + i] = '1';
            } else if (bit == '1') {
                d_output[offset + i] = '0';
            } else {
                d_output[offset + i] = bit; // Just in case of invalid input
            }
        }
    }
}

int main() {
    int N;
    printf("Enter the number of binary numbers: ");
    scanf("%d", &N);
    getchar(); // to consume newline after scanf

    const int maxLen = MAX_LEN + 1;  // +1 for null terminator
    char *h_input = new char[N * maxLen];
    char *h_output = new char[N * maxLen];

    printf("Enter %d binary strings:\n", N);
    for (int i = 0; i < N; i++) {
        char temp[MAX_LEN + 1];
        fgets(temp, maxLen, stdin);
        temp[strcspn(temp, "\n")] = '\0'; // remove newline
        strncpy(&h_input[i * maxLen], temp, maxLen);
    }

    char *d_input, *d_output;
    hipMalloc(&d_input, N * maxLen);
    hipMalloc(&d_output, N * maxLen);

    hipMemcpy(d_input, h_input, N * maxLen, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    onesComplementKernel<<<blocks, threads>>>(d_input, d_output, N, maxLen);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, N * maxLen, hipMemcpyDeviceToHost);

    printf("\nOne's Complement:\n");
    for (int i = 0; i < N; i++) {
        printf("%s -> %s\n", &h_input[i * maxLen], &h_output[i * maxLen]);
    }

    // Free memory
    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
