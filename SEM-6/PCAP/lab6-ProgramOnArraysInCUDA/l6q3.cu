// Exercise 3 - Write a program in CUDA to perform odd even transposition sort in parallel

#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void oddEvenKernel(int *d_arr, int n, int phase) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i = phase % 2 == 0 ? 2 * idx : 2 * idx + 1;

    if (i + 1 < n) {
        if (d_arr[i] > d_arr[i + 1]) {
            // Swap elements
            int temp = d_arr[i];
            d_arr[i] = d_arr[i + 1];
            d_arr[i + 1] = temp;
        }
    }
}

int main() {
    const int n = 10;
    int h_arr[n] = {7,5,4,8,9,1,2,3,6,2};
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);
    
    for (int phase = 0; phase < n; phase++) {
        oddEvenKernel<<<ceil((float)n/512), 512>>>(d_arr, n, phase);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Sorted array: ");
    for (int i = 0; i < n; i++) printf("%d ", h_arr[i]);
    hipFree(d_arr);
    return 0;
}
