#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
void intswap(int &A, int &B) {
    int temp = A;
    A = B;
    B = temp;
}

__global__ void sort(int *c, int *count) {
    int l;
    if (*count % 2 == 0)
        l = *count / 2;
    else
        l = (*count / 2) + 1;

    for (int i = 0; i < l; i++) {
        if (threadIdx.x < (*count - 1)) {
            if (c[threadIdx.x] > c[threadIdx.x + 1])
                intswap(c[threadIdx.x], c[threadIdx.x + 1]);
        }

        if (threadIdx.x < (*count - 1)) {
            if (c[threadIdx.x] > c[threadIdx.x + 1])
                intswap(c[threadIdx.x], c[threadIdx.x + 1]);
        }

    } 
}

int main() {
    int *a, *b;
    int n;
    printf("Enter size of array: ");
    scanf("%d", &n);
    int N = sizeof(int) * n;
    a = (int *)malloc(N);
    b = (int *)malloc(N);
    printf("Enter the elements of array: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &a[i]);
    }

    int *d_A, *count;
    hipMalloc(&d_A, N);
    hipMalloc((void **)&count, sizeof(int));
    hipMemcpy(d_A, a, N, hipMemcpyHostToDevice);
    hipMemcpy(count, &n, sizeof(int), hipMemcpyHostToDevice);
    
    sort<<<1, n>>>(d_A, count);
    
    hipMemcpy(b, d_A, N, hipMemcpyDeviceToHost); 
    printf("\nSORTED ARRAY: \n");
    for (int i = 0; i < n; i++) {
        printf("%d ", b[i]);
    }

    printf("\n");

    free(a);
    free(b);
    hipFree(d_A);
    hipFree(count);

    return 0;
}
