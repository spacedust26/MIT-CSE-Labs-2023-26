// Lab Exercises 1 - Write a program in CUDA to add two vectors of length N using a) block size as N b) N threads

#include "hip/hip_runtime.h"

#include<stdio.h>

const int n = 10;

__global__ void vector_additionNblocks(int *a, int *b, int *c){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n) c[tid] = a[tid] + b[tid];
}

__global__ void vector_additionNthreads(int *a, int *b, int *c){
    int tid = threadIdx.x;
    if(tid < n) c[tid] = a[tid] + b[tid];
}

int main(){
    int *d_a, *d_b, *d_c; //device copies
    int size = sizeof(int);
    
    //allocate space for device copies
    hipMalloc((void**)&d_a, n * size);
    hipMalloc((void**)&d_b, n * size);
    hipMalloc((void**)&d_c, n * size);

    //setup input values for host copies
    int a[n] = {1,2,3,4,5,6,7,8,9,0};
    int b[n] = {0,9,8,7,6,5,4,3,2,1};
    int c[n];

    //copy inputs to device
    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * size, hipMemcpyHostToDevice);

    //launch add() kernal on GPU for block size as N
    vector_additionNblocks<<<n,1>>>(d_a, d_b, d_c);

    //copy results back to host
    hipMemcpy(c, d_c, n * size, hipMemcpyDeviceToHost);

    //printing the array
    printf("The array after vector addition using N blocks is: ");
    for(int i = 0; i< n; i++){
        printf("%d ", c[i]);
    }
    printf("\n");

    //launch add() kernal on GPU for N threads
    vector_additionNthreads<<<1,n>>>(d_a, d_b, d_c); 

    //copy results back to host
    hipMemcpy(c, d_c, n * size, hipMemcpyDeviceToHost);

    //printing the array
    printf("The array after vector addition using N threads is: ");
    for(int i = 0; i< n; i++){
        printf("%d ", c[i]);
    }
    printf("\n");

    //cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}