// Solved Exercise - Program to add 2 numbers
#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *d_a, int *d_b, int *d_c){
    *d_c = *d_a + *d_b;
}

int main(){
    int h_a = 5, h_b = 3, h_c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    // Allocate memory and copy contents
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    hipMemcpy(d_a, &h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, size, hipMemcpyHostToDevice);
    
    // Launch kernerl
    add<<<1,1>>>(d_a, d_b, d_c);

    // Result display
    hipMemcpy(&h_c, d_c, size, hipMemcpyDeviceToHost);
    printf("Addition result : %d", h_c);

    // Cleanup 
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
