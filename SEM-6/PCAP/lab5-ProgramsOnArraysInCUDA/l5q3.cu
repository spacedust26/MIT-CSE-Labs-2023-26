// Lab Exercises 3 - Write a program in CUDA to process a 1D array containing angles in radian to generate sine of the angles in the output array. Use appropriate function.

#include "hip/hip_runtime.h"

#include<stdio.h>
#define PI 3.14159265359

const int n = 6;

__global__ void angles_to_sine(double *a, double *b){
    int tid = threadIdx.x;
    if(tid < n) {
        b[tid] = sinf(a[tid]);
    }
}

int main(){
    double *d_a, *d_b; //device copies
    int size = sizeof(double);
    
    //allocate space for device copies
    hipMalloc((void**)&d_a, n * size);
    hipMalloc((void**)&d_b, n * size);

    //setup input values for host copies
    double a[n] = {0, PI/6, PI/4, PI/3, PI/2, PI};
    double b[n];

    //copy inputs to device
    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);

    //launch add() kernal on GPU for N threads
    angles_to_sine<<<1,n>>>(d_a, d_b);

    //copy results back to host
    hipMemcpy(b, d_b, n * size, hipMemcpyDeviceToHost);

    //printing the array
    printf("The 1D array containing sine of angles is: \n");
    for(int i = 0; i < n; i++){
        printf("sin(%.2lf) = %.2lf \n",a[i],b[i]);
    }
    printf("\n");

    //cleanup
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}