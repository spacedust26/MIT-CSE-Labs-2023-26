// Additional Exercise - Write a program in CUDA to perform linear algebra function of the form y = ax+y where x and y are vectors and a is a scalar value

#include <stdio.h>
#include "hip/hip_runtime.h"


const int n = 5;

__global__ void linearfunction(int *x, int *y, int *a){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid < n) y[tid] = *a * x[tid] + y[tid];
}

int main(){
    int x[n], y[n], a;
    printf("Enter %d elements of vector x: ", n);
    for(int i=0;i<n;i++) scanf("%d", &x[i]);
    printf("Enter %d elements of vector y: ", n);
    for(int i=0;i<n;i++) scanf("%d", &y[i]);
    printf("Enter value of scalar a: ");
    scanf("%d", &a);

    // int x[n]= {1,2,3,4,5};
    // int y[n]= {7,2,3,8,5};
    // int a = 2;

    int *d_x, *d_y, *d_a;
    hipMalloc((void**)&d_x, n * sizeof(int));
    hipMalloc((void**)&d_y, n * sizeof(int));
    hipMalloc((void**)&d_a, sizeof(int));
    hipMemcpy(d_x, x, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);

    linearfunction<<<n, 1>>>(d_x, d_y, d_a);
    hipMemcpy(y, d_y, n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant vector: ");
    for(int i=0;i<n;i++) printf("%d ", y[i]);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_a);
    return 0;
}