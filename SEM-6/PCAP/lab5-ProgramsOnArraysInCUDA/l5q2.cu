// Lab Exercise 2 - Implement a CUDA progarm to add two vectors of length N by keeping the number of threads per block as 256(constant) and vary the number of blocks to handle N elements

#include "hip/hip_runtime.h"

#include<stdio.h>

const int n = 1000;

__global__ void vector_addition(int *a, int *b, int *c){
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid < n) c[tid] = a[tid] + b[tid];
}

int main(){
    int *d_a, *d_b, *d_c; //device copies
    int size = sizeof(int);
    
    //allocate space for device copies
    hipMalloc((void**)&d_a, n * size);
    hipMalloc((void**)&d_b, n * size);
    hipMalloc((void**)&d_c, n * size);

    //setup input values for host copies
    // int a[n] = {1,2,3,4,5,6,7,8,9,0};
    // int b[n] = {0,9,8,7,6,5,4,3,2,1};
    // int c[n];
    int a[n], b[n], c[n];
    for(int i = 0 ; i < n ; i++){
        a[i] = i;
        b[i] = i + 10;
    }

    //copy inputs to device
    hipMemcpy(d_a, a, n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * size, hipMemcpyHostToDevice);

    //launch add() kernal on GPU for N threads
    vector_addition<<<ceil((float)n/256),256>>>(d_a, d_b, d_c); //1 

    //copy results back to host
    hipMemcpy(c, d_c, n * size, hipMemcpyDeviceToHost);

    //printing the array
    printf("The array after vector addition using N threads is: ");
    for(int i = 0; i< n; i++){
        printf("%d ", c[i]);
    }
    printf("\n");

    //cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}