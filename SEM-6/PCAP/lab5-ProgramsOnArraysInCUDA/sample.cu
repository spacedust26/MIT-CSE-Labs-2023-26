 //Solved Exercise: Program to add two numbers
 
 #include "hip/hip_runtime.h"
 
 #include<stdio.h>

__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

int main(){
    int a,b,c; //host copies
    int *d_a, *d_b, *d_c; //device copies
    int size = sizeof(int);
    
    //allocate space for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    //setup input values
    a=5, b = 5;;

    //copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    //launch add() kernal on GPU
    add<<<1,1>>>(d_a, d_b, d_c); // 1 block 1 thread

    //copy results back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result : %d\n", c);

    //cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
