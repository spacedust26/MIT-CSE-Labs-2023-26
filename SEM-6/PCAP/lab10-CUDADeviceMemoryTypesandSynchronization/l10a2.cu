// Additional exercise 2 - write a program in CUDA to perform tiled 1D convolution operation on the input array N of size width using the mask array, M of size mask_width to produce the output array, P of size width

#include <stdio.h>
#include <hip/hip_runtime.h>

#define MASK_WIDTH 3
#define TILE_WIDTH 8
#define RADIUS (MASK_WIDTH / 2)

// Constant memory for mask
__constant__ float M[MASK_WIDTH];

// 1D Convolution kernel using shared memory
__global__ void convolution1D(float* N, float* P, int width) {
    __shared__ float N_s[TILE_WIDTH + MASK_WIDTH - 1];

    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_idx = threadIdx.x + RADIUS;

    // Load shared memory with halo
    if (global_idx < width)
        N_s[local_idx] = N[global_idx];
    else
        N_s[local_idx] = 0.0f;

    if (threadIdx.x < RADIUS) {
        // Left halo
        int left_idx = global_idx - RADIUS;
        N_s[threadIdx.x] = (left_idx >= 0) ? N[left_idx] : 0.0f;

        // Right halo
        int right_idx = global_idx + TILE_WIDTH;
        if (threadIdx.x + TILE_WIDTH < TILE_WIDTH + RADIUS)
            N_s[threadIdx.x + TILE_WIDTH + RADIUS] = (right_idx < width) ? N[right_idx] : 0.0f;
    }

    __syncthreads();

    // Perform convolution
    if (global_idx < width) {
        float sum = 0.0f;
        for (int j = 0; j < MASK_WIDTH; j++) {
            sum += M[j] * N_s[local_idx - RADIUS + j];
        }
        P[global_idx] = sum;
    }
}

int main() {
    int width = 16;

    float h_N[16] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    float h_M[MASK_WIDTH] = {0.25, 0.5, 0.25}; // Sample blur kernel
    float h_P[16];

    float *d_N, *d_P;

    // Allocate device memory
    hipMalloc((void**)&d_N, width * sizeof(float));
    hipMalloc((void**)&d_P, width * sizeof(float));

    // Copy data to device
    hipMemcpy(d_N, h_N, width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(M), h_M, MASK_WIDTH * sizeof(float));

    // Kernel launch
    int numBlocks = (width + TILE_WIDTH - 1) / TILE_WIDTH;
    convolution1D<<<numBlocks, TILE_WIDTH>>>(d_N, d_P, width);

    // Copy result back to host
    hipMemcpy(h_P, d_P, width * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Input:\n");
    for (int i = 0; i < width; i++) {
        printf("%.1f ", h_N[i]);
    }

    printf("\n\nOutput (Convolved):\n");
    for (int i = 0; i < width; i++) {
        printf("%.2f ", h_P[i]);
    }
    printf("\n");

    // Free memory
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}
