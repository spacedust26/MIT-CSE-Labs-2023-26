#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 256 

__global__ void inclusiveScanKernel(int *d_out, int *d_in, int n) {
    __shared__ int temp[BLOCK_SIZE];  // Shared memory for block scan
    int thid = threadIdx.x;
    // Load input into shared memory
    int ai = thid;
    if (ai < n) temp[ai] = d_in[ai];
    else temp[ai] = 0;
    __syncthreads();
    // Up-sweep (reduction phase)
    for (int d = 1; d < BLOCK_SIZE; d *= 2) {
        if (thid >= d)  temp[thid] += temp[thid - d];
        __syncthreads();
    }
    // Write results to global memory
    if (thid < n) d_out[thid] = temp[thid];
}

void inclusiveScan(int *h_out, int *h_in, int n) {
    int *d_in, *d_out;
    size_t size = n * sizeof(int);
    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
    int blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    inclusiveScanKernel<<<blocks, BLOCK_SIZE>>>(d_out, d_in, n);
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}

int main() {
    int n = 10;
    int h_in[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    int h_out[10];
    inclusiveScan(h_out, h_in, n);
    printf("Inclusive Scan Result:\n");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_out[i]);
    }
    printf("\n");
    return 0;
}