// Lab exercise 3 - Write a program in CUDA to perform incluisve scan algorithm

#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16  // Size of input array (can change as needed)

__global__ void inclusiveScanKernel(int *input, int *output, int n) {
    __shared__ int temp[N];
    int tid = threadIdx.x;

    if (tid < n)
        temp[tid] = input[tid];
    __syncthreads();

    // Inclusive scan using Hillis-Steele algorithm
    for (int offset = 1; offset < n; offset *= 2) {
        int val = 0;
        if (tid >= offset)
            val = temp[tid - offset];
        __syncthreads();
        if (tid < n)
            temp[tid] += val;
        __syncthreads();
    }

    if (tid < n)
        output[tid] = temp[tid];
}

int main() {
    int h_input[N], h_output[N];
    int *d_input, *d_output;

    // Initialize input array
    printf("Input array:\n");
    for (int i = 0; i < N; i++) {
        h_input[i] = i + 1;  // Example: 1, 2, 3, ..., N
        printf("%d ", h_input[i]);
    }
    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_output, N * sizeof(int));
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    inclusiveScanKernel<<<1, N>>>(d_input, d_output, N);
    hipMemcpy(h_output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("Inclusive scan output:\n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_output[i]);
    }
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
