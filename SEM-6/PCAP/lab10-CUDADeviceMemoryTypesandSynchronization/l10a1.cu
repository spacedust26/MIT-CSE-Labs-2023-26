// Additional exercise 1 - Write a program in CUDA which displays a shopping mall item menu with its price, The N number of friens are allowed to purchse as many items as they want. Calculate the total purchase done by N freinds

#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_ITEMS 5
#define NUM_FRIENDS 3

// CUDA kernel to calculate total purchase for each friend
__global__ void calculateTotal(float* itemPrices, int* quantities, float* totals, int numItems) {
    int friendId = blockIdx.x;
    float sum = 0.0f;

    for (int i = 0; i < numItems; i++) {
        sum += itemPrices[i] * quantities[friendId * numItems + i];
    }

    totals[friendId] = sum;
}

int main() {
    // Item names and prices
    const char* itemNames[NUM_ITEMS] = {"Shoes", "Shirt", "Jeans", "Watch", "Bag"};
    float itemPrices[NUM_ITEMS] = {50.0f, 30.0f, 60.0f, 120.0f, 40.0f};

    // Quantities each friend purchased [NUM_FRIENDS x NUM_ITEMS]
    int quantities[NUM_FRIENDS][NUM_ITEMS] = {
        {1, 2, 0, 0, 1}, // Friend 1
        {0, 1, 1, 0, 0}, // Friend 2
        {2, 0, 1, 1, 0}  // Friend 3
    };

    // Device pointers
    float *d_itemPrices, *d_totals;
    int *d_quantities;

    // Allocate memory on device
    hipMalloc((void**)&d_itemPrices, NUM_ITEMS * sizeof(float));
    hipMalloc((void**)&d_quantities, NUM_FRIENDS * NUM_ITEMS * sizeof(int));
    hipMalloc((void**)&d_totals, NUM_FRIENDS * sizeof(float));

    // Copy data to device
    hipMemcpy(d_itemPrices, itemPrices, NUM_ITEMS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_quantities, quantities, NUM_FRIENDS * NUM_ITEMS * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel: 1 block per friend
    calculateTotal<<<NUM_FRIENDS, 1>>>(d_itemPrices, d_quantities, d_totals, NUM_ITEMS);

    // Copy totals back to host
    float totals[NUM_FRIENDS];
    hipMemcpy(totals, d_totals, NUM_FRIENDS * sizeof(float), hipMemcpyDeviceToHost);

    // Print menu
    printf("\n--- Shopping Mall Menu ---\n");
    for (int i = 0; i < NUM_ITEMS; i++) {
        printf("%s: $%.2f\n", itemNames[i], itemPrices[i]);
    }

    // Print individual and grand total
    printf("\n--- Friends' Purchases ---\n");
    float grandTotal = 0.0f;
    for (int i = 0; i < NUM_FRIENDS; i++) {
        printf("Friend %d total: $%.2f\n", i + 1, totals[i]);
        grandTotal += totals[i];
    }

    printf("\nTotal purchase by all friends: $%.2f\n", grandTotal);

    // Free device memory
    hipFree(d_itemPrices);
    hipFree(d_quantities);
    hipFree(d_totals);

    return 0;
}
