// Lab Exercise 1 - Write a program in CUDA to perform parallele Sparse Matrix - Vector multiplication using compressed sparse row (CSR) storage format. Represent the input sparse matrix in CSR format in the host code
 
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
 
__global__ void csr_kernel(int num_rows, int *row_ptr, int *col_index, int *data, int *x, int *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        int dot = 0;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        for (int i = row_start; i < row_end; i++) {
            dot += data[i] * x[col_index[i]];
        }
        y[row] = dot;
    }
}
 
int main(){
    int num_rows, num_cols;
    printf("Enter number of rows and columns: ");
    scanf("%d %d", &num_rows, &num_cols);
    int matrix[num_rows][num_cols];
    int num_non_zeros = 0;
    printf("Enter the sparse matrix (%dx%d):\n", num_rows, num_cols);
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            scanf("%d", &matrix[i][j]);
            if (matrix[i][j] != 0) num_non_zeros++;
        }
    }
 
    int data[num_non_zeros];
    int col_index[num_non_zeros];
    int row_ptr[num_rows + 1];
 
    // Convert matrix to CSR format
    row_ptr[0] = 0;
    int idx = 0;
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_cols; j++) {
            if (matrix[i][j] != 0) {
                data[idx] = matrix[i][j];
                col_index[idx] = j;
                idx++;
            }
        }
        row_ptr[i + 1] = idx;
    }
    int x[num_cols];
    printf("Enter vector x (%d values):\n", num_cols);
    for (int i = 0; i < num_cols; i++) scanf("%d", &x[i]);
    int y[num_rows];
    int *d_row_ptr, *d_col_index;
    int *d_data, *d_x, *d_y;
    hipMalloc((void**)&d_row_ptr, (num_rows + 1) * sizeof(int));
    hipMalloc((void**)&d_col_index, num_non_zeros * sizeof(int));
    hipMalloc((void**)&d_data, num_non_zeros * sizeof(int));
    hipMalloc((void**)&d_x, num_cols * sizeof(int));
    hipMalloc((void**)&d_y, num_rows * sizeof(int));
    hipMemcpy(d_row_ptr, row_ptr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, col_index, num_non_zeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, data, num_non_zeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_cols * sizeof(int), hipMemcpyHostToDevice);
    int block_size = 256;
    int grid_size = (num_rows + block_size - 1) / block_size;
    csr_kernel<<<grid_size, block_size>>>(num_rows, d_row_ptr, d_col_index, d_data, d_x, d_y);
    hipMemcpy(y, d_y, num_rows * sizeof(int), hipMemcpyDeviceToHost);
 
    printf("CSR Representation:\n");
    printf("Data Array: ");
    for (int i = 0; i < num_non_zeros; i++) printf("%d ", data[i]);
    printf("\nCol_index Array: ");
    for (int i = 0; i < num_non_zeros; i++) printf("%d ", col_index[i]);
    printf("\nRow_ptr Array: ");
    for (int i = 0; i <= num_rows; i++) printf("%d ", row_ptr[i]);
    printf("\n");
    printf("\nResult vector y:\n");
    for (int i = 0; i < num_rows; i++)printf("%d\n", y[i]);
 
    hipFree(d_row_ptr);
    hipFree(d_col_index);
    hipFree(d_data);
    hipFree(d_x);
    hipFree(d_y);
    return 0;
}
 