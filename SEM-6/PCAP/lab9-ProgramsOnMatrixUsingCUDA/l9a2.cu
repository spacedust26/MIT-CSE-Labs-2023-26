// Additional Exercise 2 - Write a CUDA program that reads a character type matrix A and integer type matrix B of size M X N. It produces an output string STR such that, every character of A is repeated r times (where r is the integer value in matrix B which is having the same index as that of the character taken in A). Write the kernel such that every value of input matrix must be produced required number of times by one thread.
// Example : A = [[p, C, a, P][e, X, a, M]]
// B = [[1, 2, 4, 3][2, 4, 3, 2]]
// Output String STR: pCCaaaaPPPeeXXXXaaaMM

#include <stdio.h>
#include <hip/hip_runtime.h>


#define M 2
#define N 4
#define MAX_LEN 1000 

__device__ int getOffset(int* B, int row, int col, int rows, int cols) {
    int offset = 0;
    for (int i = 0; i < row; i++)
        for (int j = 0; j < cols; j++)
            offset += B[i * cols + j];
    for (int j = 0; j < col; j++)
        offset += B[row * cols + j];
    return offset;
}
__global__ void repeatChars(char* A, int* B, char* STR, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        char ch = A[idx];
        int r = B[idx];
        int offset = getOffset(B, row, col, rows, cols);
        for (int i = 0; i < r; i++) {
            STR[offset + i] = ch;
        }
    }
}

int main() {
    char A[M][N] = {
        {'p', 'C', 'a', 'P'},
        {'e', 'X', 'a', 'M'}
    };

    int B[M][N] = {
        {1, 2, 4, 3},
        {2, 4, 3, 2}
    };
    char flatA[M * N];
    int flatB[M * N];
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++) {
            flatA[i * N + j] = A[i][j];
            flatB[i * N + j] = B[i][j];
        }

    int totalLen = 0;
    for (int i = 0; i < M * N; i++) {
        totalLen += flatB[i];
    }

    char *d_A, *d_STR;
    int *d_B;
    hipMalloc((void**)&d_A, M * N * sizeof(char));
    hipMalloc((void**)&d_B, M * N * sizeof(int));
    hipMalloc((void**)&d_STR, totalLen * sizeof(char));
    hipMemcpy(d_A, flatA, M * N * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_B, flatB, M * N * sizeof(int), hipMemcpyHostToDevice);
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (M + 15) / 16);
    repeatChars<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_STR, M, N);
    char STR[MAX_LEN] = {0};
    hipMemcpy(STR, d_STR, totalLen * sizeof(char), hipMemcpyDeviceToHost);
    STR[totalLen] = '\0';
    printf("Output STR: %s\n", STR);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_STR);
    return 0;
}
