// Additional Exercise 1 - Write a CUDA program which reads an input matrix A of size M X N and produces an output matrix B of size M X N such that, each element of the output matrix is calculated in parallel. Each element, B[i][j], in the output matrix is obtained by adding the elements in ith row and jth column of the input matrix A.
// Example A = [[1, 2, 3], [4, 5, 6]]
// Output B = [[11, 13, 15], [20, 22, 24]]

#include <stdio.h>
#include <hip/hip_runtime.h>


#define M 2
#define N 3
__global__ void computeMatrix(int* A, int* B, int* rowSum, int* colSum, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows && col < cols) {
        int idx = row * cols + col;
        B[idx] = rowSum[row] + colSum[col];
    }
}

int main() {
    int A[M][N] = {
        {1, 2, 3},
        {4, 5, 6}
    };
    int B[M][N];
    int rowSum[M] = {0};
    int colSum[N] = {0};

    for (int i = 0; i < M; ++i)
        for (int j = 0; j < N; ++j) {
            rowSum[i] += A[i][j];
            colSum[j] += A[i][j];
        }

    int size = M * N * sizeof(int);
    int *d_A, *d_B, *d_rowSum, *d_colSum;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_rowSum, M * sizeof(int));
    hipMalloc((void**)&d_colSum, N * sizeof(int));

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_rowSum, rowSum, M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colSum, colSum, N * sizeof(int), hipMemcpyHostToDevice);
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (M + 15) / 16);

    computeMatrix<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_rowSum, d_colSum, M, N);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);

    printf("Output Matrix B:\n");
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j)
            printf("%d ", B[i][j]);
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_rowSum);
    hipFree(d_colSum);

    return 0;
}
