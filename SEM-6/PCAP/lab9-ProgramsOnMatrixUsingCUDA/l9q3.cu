// Lab Exercise 3 - Write a CUDA program that reads a matrix A of size M X N and produce an output matrix B of the same size such that it replaces all the non border elements (numbers in bold) of A with its equivalent 1's complement and remaining elements same as matrix A.
 
#include<stdio.h>
#include <hip/hip_runtime.h>
 
__global__ void processMatrix(int *A, int *B, int M, int N) {
    int row = threadIdx.y;
    int col = threadIdx.x;
    if (row < M && col < N) {
        int index = row * N + col;
        if (row == 0 || row == M - 1 || col == 0 || col == N - 1) {
            B[index] = A[index];
        } else {
            B[index] = ~A[index];
        }
    }
}
 
void printBinary(int num) {
    for (int i = 2; i >= 0; i--) {
        printf("%d", (num >> i) & 1);
    }
}
 
int main() {
    int M, N;
    printf("Enter M and N: ");
    scanf("%d %d", &M, &N);
 
    int size = M * N * sizeof(int);
    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
 
    printf("Enter matrix elements:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            scanf("%d", &h_A[i * N + j]);
        }
    }
 
    int *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
 
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    dim3 threadsPerBlock(N, M);
    processMatrix<<<1, threadsPerBlock>>>(d_A, d_B, M, N);
    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
 
    printf("Result (in binary):\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            if (i == 0 || i == M - 1 || j == 0 || j == N - 1) printf("%d", h_B[i * N  + j]);
            else printBinary(h_B[i * N + j]);
            printf(" ");
        }
        printf("\n");
    }
 
    free(h_A);
    free(h_B);
    hipFree(d_A);
    hipFree(d_B);
 
    return 0;
}