// Lab Exercide 2 - Write a program in CUDA to read M X N matrix A and replace 1st row of this matrix by same elements, 2nd row elements by square of each element and 3rd row elements by cube of each element and so on.
 
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
 
__global__ void transformMatrix(int *A, int M, int N) {
    int row = threadIdx.x;
    if (row < M) {
        for (int col=0; col<N; col++) {
            A[row * N + col] = pow(A[row * N + col],(row + 1));
        }
    }
}
 
int main() {
    int M, N;
    printf("Enter M and N: ");
    scanf("%d %d", &M, &N);
    int size = M * N * sizeof(int);
    int *h_A = (int*)malloc(size);
    printf("Enter matrix elements:\n");
    for (int i = 0; i < M * N; i++) {
        scanf("%d", &h_A[i]);
    }
    int *d_A;
    hipMalloc(&d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    transformMatrix<<<1, M>>>(d_A, M, N);
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    printf("Result:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_A[i * N + j]);
        }
        printf("\n");
    }
    free(h_A);
    hipFree(d_A);
    return 0;
}